#include <stdexcept>

#include "solver.cuh"

Solver::Solver(NeuralNet *model, void *X_train, int *y_train, void *X_val,
               int *y_val, int num_epoch, UpdateRule update_rule,
               double learning_rate, double learning_rate_decay, int num_train,
               int num_val) {
  if ((model->batch_size == 0) || (model->num_layers == 0))
    throw std::invalid_argument(
        "Model is not initialized. Use parameterized constructor.");
  this->model = model;
  this->X_train = X_train, this->X_val = X_val;
  this->y_train = y_train, this->y_val = y_val;
  this->num_epoch = num_epoch;
  this->update_rule = update_rule;
  this->learning_rate = learning_rate,
  this->learning_rate_decay = learning_rate_decay;

  this->num_train = num_train, this->num_val = num_val;
  this->num_features = model->input_channels * model->input_h * model->input_w;

  hipEventCreate(&start);
  hipEventCreate(&stop);
}

float Solver::step(int start_X, int start_y, int *correct_count, bool train,
                   bool doo) {
  std::vector<float> t1, t2;
  return this->step(start_X, start_y, t1, t2, correct_count, train, doo);
}

float Solver::step(int start_X, int start_y, std::vector<float> &fwd_dnn_lag,
                   std::vector<float> &bwd_dnn_lag, int *correct_count,
                   bool train, bool doo) {
  float temp_loss;

  if (model->data_type == HIPDNN_DATA_FLOAT)
    model->getLoss(&(((float *)X_train)[start_X]), &y_train[start_y],
                   learning_rate, fwd_dnn_lag, bwd_dnn_lag, train,
                   correct_count, &temp_loss, doo);
  else if (model->data_type == HIPDNN_DATA_DOUBLE)
    model->getLoss(&(((double *)X_train)[start_X]), &y_train[start_y],
                   learning_rate, fwd_dnn_lag, bwd_dnn_lag, train,
                   correct_count, &temp_loss, doo);

  hipDeviceSynchronize();
  return temp_loss;
}

void Solver::train(std::vector<float> &loss, std::vector<int> &val_acc,
                   std::vector<float> &batch_times, bool doo) {
  int batch_size = model->batch_size;
  int num_train_batches = num_train / model->batch_size;
  int num_val_batches = num_val / model->batch_size;
  for (int i = 0; i < num_epoch; i++) {
    std::cout << "Epoch " << i << std::endl;
    for (int j = 0; j < num_train_batches; j++) {
      int start_sample = j * num_features * batch_size;

      float milli = 0;
      hipEventRecord(start, model->stream_compute);

      float temp_loss = step(start_sample, j * batch_size, NULL, true, doo);

      hipEventRecord(stop, model->stream_compute);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milli, start, stop);
      std::cout << "Batch: " << j
                << " One forward, backward pass time(ms): " << milli
                << std::endl;

      loss.push_back(temp_loss);
      batch_times.push_back(milli);
    }
    std::cout << "LOSS: " << loss[loss.size() - 1] << std::endl;

    int correct_count = 0;
    for (int j = 0; j < num_val_batches; j++) {
      int start_sample = j * num_features * batch_size;
      int temp_correct_count = 0;
      float temp_loss =
          step(start_sample, j * batch_size, &temp_correct_count, false, doo);

      correct_count += temp_correct_count;
    }
    val_acc.push_back(correct_count);
    std::cout << "VAL_ACC: " << val_acc[i] << std::endl;
    learning_rate *= learning_rate_decay;
    std::cout << "learning_rate: " << learning_rate << std::endl;
  }
}

void Solver::checkAccuracy(void *X, int *y, int num_samples, int *num_correct) {
  int batch_size = model->batch_size;
  int num_iter = num_samples / batch_size;
  *num_correct = 0;
  for (int i = 0; i < num_iter; i++) {
    int start_sample = i * num_features * batch_size;
    int temp_correct_count;
    if (model->data_type == HIPDNN_DATA_FLOAT)
      model->getLoss(&(((float *)X)[start_sample]), &y[i * batch_size],
                     learning_rate, false, &temp_correct_count, NULL, false);
    else if (model->data_type == HIPDNN_DATA_DOUBLE)
      model->getLoss(&(((double *)X)[start_sample]), &y[i * batch_size],
                     learning_rate, false, &temp_correct_count, NULL, false);
    *num_correct = *num_correct + temp_correct_count;
  }
}
