#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hipDNN.h>
#include <hiprand.h>
#include <time.h>

#include <cstdio>
#include <string>

#include "neural_net.cuh"

template <typename T>
__global__ void softmaxLossBackProp(int *y, T *SO, T *dSO, int batch_size,
                                    int output_size, float eps) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= batch_size) return;
  int cur_class = static_cast<int>(y[i]);
  dSO[i * output_size + cur_class] =
      -1 / (SO[i * output_size + cur_class] * batch_size + eps);
}

void NeuralNet::getLoss(void *X, int *y, double learning_rate, bool train,
                        int *correct_count, float *loss, bool doo) {
  std::vector<float> t1, t2;
  this->getLoss(X, y, learning_rate, t1, t2, train, correct_count, loss, doo);
}

void NeuralNet::getLoss(void *X, int *y, double learning_rate,
                        std::vector<float> &fwd_dnn_lag,
                        std::vector<float> &bwd_dnn_lag, bool train,
                        int *correct_count, float *scalar_loss, bool doo) {
  hipMemGetInfo(&free_bytes, &total_bytes);
  int bef0 = free_bytes;
  hipMalloc(&layer_input[0], layer_input_size[0] * data_type_size);
  hipMemGetInfo(&free_bytes, &total_bytes);
  int aft0 = free_bytes;
  std::cout << "Allocated to layer 0: " << (bef0 - aft0)
            << " free: " << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";
  hipMemcpy(layer_input[0], X,
             batch_size * input_channels * input_h * input_w * data_type_size,
             hipMemcpyHostToDevice);
  if (train == true)
    hipMemcpy(this->y, y, batch_size * data_type_size, hipMemcpyHostToDevice);

  float alpha = 1.0, beta = 0.0;
  float Salpha = 1.0, Sbeta = 0.0;
  double Dalpha = 1.0, Dbeta = 0.0;

  // Display layer_input_size in bytes
  for (int c = 0; c < num_layers; c++)
    std::cout << "layer_input_size[" << c
              << "] = " << layer_input_size[c] * data_type_size << std::endl;

  /************************ Forward Propagation starts ***********************/
  std::cout << "Forward Propagation starts: " << '\n';
  size_t buffer_bytes = 1024 * 1024 * 1024;  // 2GB
  int ttl_allocated = 0;
  std::vector<int> free_layer;  // Which layers to free
  for (int i = 0; i < num_layers; i++) {
    if (train == false && i == num_layers - 1) break;

    hipMemGetInfo(&free_bytes, &total_bytes);
    int bef = free_bytes;
    hipMalloc(&layer_input[i + 1], layer_input_size[i + 1] * data_type_size);
    hipMemGetInfo(&free_bytes, &total_bytes);
    int aft = free_bytes;
    ttl_allocated += (bef - aft);
    std::cout << "Allocated to layer " << i + 1 << ": " << (bef - aft)
              << " free: " << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";

    // Push the layer_input_size + weights_size to the heap of ith layer
    if (i > 0) {
      layer_input_pq.push({layer_input_size[i], i});
      std::cout << "Layer inserted: " << i << "\n";
    }

    hipMemGetInfo(&free_bytes, &total_bytes);
    std::cout << "Before Offload and computation of layer " << i << " : "
              << free_bytes / (1024.0 * 1024.0 * 1024.0) << '\n';

    size_t temp_free_bytes = free_bytes;                  // Current free bytes
    size_t free_memory = temp_free_bytes - buffer_bytes - buffer_bytes;  // Free memory made 2 GB reserved
    size_t layer_size =
        layer_input_size[i + 2] * data_type_size;  // Size of the layer

    // Decrement free_memory by i+2 th layer fwd workspace size.
    // This make sures that i+2 the layer can be allocated smoothly when
    // needed
    if ((i + 2 < num_layers) && (layer_type[i + 2] == CONV)) {
      ConvLayerParams *cur_params = (ConvLayerParams *)params[i + 2];
      layer_size += cur_params->fwd_workspace_size;
    }

    if ((i + 2 < num_layers) && (free_memory <= layer_size)) {
      std::cout << "GPU memory is low, offloading to CPU" << std::endl;
      std::cout << (free_bytes - buffer_bytes - buffer_bytes) / float(buffer_bytes) << " <= "
                << layer_input_size[i + 2] * data_type_size /
                       float(buffer_bytes)
                << '\n';

      /************* Heap logic with workspace fix ********************/

      bool cond1 = (free_memory <= layer_size);
      bool cond2 = (!layer_input_pq.empty());

      // Display cond1 and cond2
      std::cout << "cond1: " << cond1 << " cond2: " << cond2 << std::endl;

      // Display cond1 && cond2
      std::cout << "Condition: " << (cond1 && cond2) << std::endl;




      // While the free memory is less than or equal to the (i+2)th layer
      // input size or the heap is not empty
      while ((free_memory <= layer_size) && (!layer_input_pq.empty())) {
        int temp = layer_input_pq.top().second;  // Get the layer index on top
                                                 // of the heap
        std::cout << "Layer to offload: " << temp << std::endl;
        std::cout << "Size of the layer to offload: "
                  << layer_input_pq.top().first * data_type_size /
                         float(buffer_bytes)
                  << std::endl;
        free_layer.push_back(temp);  // Add the layer index to the free layer
                                     // vector

        // Update the free bytes
        temp_free_bytes += layer_input_pq.top().first * data_type_size;
        std::cout << "Free gigabytes in GPU: "
                  << temp_free_bytes / float(buffer_bytes) << std::endl;
        offloaded[temp] = true;  // Mark the layer as offloaded

        // Copy the layer to host
        // Only offload layer_input and not workspace
        hipMemcpyAsync(h_layer_input[temp], layer_input[temp],
                        layer_input_size[temp] * data_type_size,
                        hipMemcpyDeviceToHost, stream_memory);
        layer_input_pq.pop();  // Remove the layer from the heap
        std::cout << "New Top: " << layer_input_pq.top().second << "\n";
        free_memory = temp_free_bytes - buffer_bytes - buffer_bytes;
      }
      /*************************************************************/
    }

    //     if(i>1 && train == true && doo==true)
    // //     {
    // //       hipMemGetInfo(&free_bytes, &total_bytes);
    // //       std::cout << "Before Offload: " << free_bytes <<'\n';
    // //       std::cout << "hipMemPrefetchAsync: " <<
    //     hipMemPrefetchAsync(layer_input[i-1],
    //     layer_input_size[i-1]*data_type_size, hipCpuDeviceId,
    //     stream_memory); //<< '\n';
    // //     }
    hipMemGetInfo(&free_bytes, &total_bytes);
    std::cout << "Before Computation of Layer " << i << ": "
              << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";
    if (layer_type[i] == CONV) {
      ConvLayerParams *cur_params = (ConvLayerParams *)params[i];

      this->workspace_size = cur_params->fwd_workspace_size;

      hipMalloc(&(this->workspace), cur_params->fwd_workspace_size);

      // Computation
      checkCUDNN(hipdnnConvolutionForward(
          cudnn_handle, &alpha, cur_params->input_tensor, layer_input[i],
          cur_params->filter_desc, cur_params->W, cur_params->conv_desc,
          cur_params->fwd_algo, this->workspace, this->workspace_size, &beta,
          cur_params->output_tensor, layer_input[i + 1]));
      hipMemGetInfo(&free_bytes, &total_bytes);
      std::cout << "After hipdnnConvolutionForward " << i << ": "
                << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";
      checkCUDNN(hipdnnAddTensor(cudnn_handle, &alpha, cur_params->bias_desc,
                                cur_params->b, &alpha,
                                cur_params->output_tensor, layer_input[i + 1]));
      hipMemGetInfo(&free_bytes, &total_bytes);
      std::cout << "After hipdnnAddTensor " << i << ": "
                << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";
      // If activation required
      if (cur_params->activation_mode != ACTIVATION_NONE) {
        checkCUDNN(hipdnnActivationForward(
            cudnn_handle, cur_params->actv_desc, &alpha,
            cur_params->output_tensor, layer_input[i + 1], &beta,
            cur_params->output_tensor, layer_input[i + 1]));
      }
      hipMemGetInfo(&free_bytes, &total_bytes);
      std::cout << "After hipdnnActivationForward " << i << ": "
                << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";
    }

    else if (layer_type[i] == FULLY_CONNECTED) {
      FCLayerParams *cur_params = (FCLayerParams *)params[i];

      if (data_type == HIPDNN_DATA_FLOAT) {
        hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, cur_params->C_out,
                    batch_size, cur_params->C_in, &Salpha,
                    (float *)cur_params->W, cur_params->C_out,
                    (float *)layer_input[i], cur_params->C_in, &Sbeta,
                    (float *)layer_input[i + 1], cur_params->C_out);
        hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, cur_params->C_out,
                    batch_size, 1, &Salpha, (float *)cur_params->b,
                    cur_params->C_out, (float *)one_vec, 1, &Salpha,
                    (float *)layer_input[i + 1], cur_params->C_out);
      } else if (data_type == HIPDNN_DATA_DOUBLE) {
        hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, cur_params->C_out,
                    batch_size, cur_params->C_in, &Dalpha,
                    (double *)cur_params->W, cur_params->C_out,
                    (double *)layer_input[i], cur_params->C_in, &Dbeta,
                    (double *)layer_input[i + 1], cur_params->C_out);
        hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, cur_params->C_out,
                    batch_size, 1, &Dalpha, (double *)cur_params->b,
                    cur_params->C_out, (double *)one_vec, 1, &Dalpha,
                    (double *)layer_input[i + 1], cur_params->C_out);
      }
      if (cur_params->activation_mode != ACTIVATION_NONE) {
        //         hipMemGetInfo(&free_bytes, &total_bytes);
        //         std::cout << "Before Offload: " << free_bytes <<'\n';
        checkCUDNN(hipdnnActivationForward(
            cudnn_handle, cur_params->actv_desc, &alpha,
            cur_params->output_tensor, layer_input[i + 1], &beta,
            cur_params->output_tensor, layer_input[i + 1]));
      }
    } else if (layer_type[i] == POOLING) {
      PoolingLayerParams *cur_params = (PoolingLayerParams *)params[i];
      checkCUDNN(
          hipdnnPoolingForward(cudnn_handle, cur_params->pool_desc, &alpha,
                              cur_params->input_tensor, layer_input[i], &beta,
                              cur_params->output_tensor, layer_input[i + 1]));
    } else if (layer_type[i] == ACTV) {
      std::cout << "Actv Layer\n";
      std::cout << "Panic!! ACTV wrong place\n";
      exit(0);
      ActivationLayerParams *cur_params = (ActivationLayerParams *)params[i];
      checkCUDNN(hipdnnActivationForward(
          cudnn_handle, cur_params->actv_desc, &alpha, cur_params->input_tensor,
          layer_input[i], &beta, cur_params->input_tensor, layer_input[i + 1]));
    }
    // else if (layer_type[i] == SOFTMAX)
    // {
    //   // std::cout << "Softmax\n";
    //   //   std::cout << "Panic!! SOFTMAX wrong place\n";
    //   //   exit(0);
    //   if (train == true)
    //   {
    //     SoftmaxLayerParams *cur_params = (SoftmaxLayerParams *)params[i];
    //     checkCUDNN(hipdnnSoftmaxForward(
    //         cudnn_handle, cur_params->algo, cur_params->mode, &alpha,
    //         cur_params->input_tensor, layer_input[i], &beta,
    //         cur_params->input_tensor, layer_input[i + 1]));
    //   }
    // }
    // synchronization
    // hipDeviceSynchronize();

    // if next layer is ACTV or SOFTMAX, complete that and come to
    // synchronization the case in above if for ACTV and SOFTMAX never occurs
    if (layer_type[i + 1] == SOFTMAX) {
      i++;
      if (train == true) {
        layer_input[i + 1] = layer_input[i];
        SoftmaxLayerParams *cur_params = (SoftmaxLayerParams *)params[i];
        checkCUDNN(hipdnnSoftmaxForward(
            cudnn_handle, cur_params->algo, cur_params->mode, &alpha,
            cur_params->input_tensor, layer_input[i], &beta,
            cur_params->input_tensor, layer_input[i + 1]));
      }
      // i--;
    }
    hipMemGetInfo(&free_bytes, &total_bytes);
    std::cout << "Before Synchronization " << i << ": "
              << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";
    hipStreamSynchronize(stream_compute);
    hipMemGetInfo(&free_bytes, &total_bytes);
    std::cout << "After Synchronization " << i << ": "
              << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";
    hipStreamSynchronize(stream_memory);
    hipMemGetInfo(&free_bytes, &total_bytes);
    std::cout << "After Computation of Layer " << i << ": "
              << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";

    /**************************** Free up memory ****************************/
    if (layer_type[i] == CONV) hipFree(this->workspace);  // free workspace

    for (int c = 0; c < free_layer.size(); c++)
      hipFree(layer_input[free_layer[c]]);  // free layer_input

    if (train == false && offloaded[i] == false) hipFree(layer_input[i]);
    free_layer.clear();  // clear free_layer
    /**********************************************************************/

    hipMemGetInfo(&free_bytes, &total_bytes);

    std::cout << "After Offload and computation of layer " << i << " : "
              << free_bytes / (1024.0 * 1024.0 * 1024.0) << '\n';
  }
  std::cout << "Forward Propagation ends: " << '\n';
  /************************ Forward Propagation ends ***********************/

  /************************ Offloaded layers Displayed ***********************/
  int flag = false;
  for (int c = 0; c < num_layers; c++)
    if (offloaded[c]) {
      flag = true;
      break;
    }
  if (flag) {
    std::cout << "\nOffloaded Layers: ";
    for (int c = 0; c < num_layers; c++)
      if (offloaded[c]) std::cout << c << " ";
  } else
    std::cout << "\nNo Offloaded Layers: ";
  std::cout << '\n';

  // Empty the priority queue
  while (!layer_input_pq.empty())
    layer_input_pq.pop();
  /***************************************************************************/

  /************************** Accuracy Computation **************************/
  if (train == false) {
    compareOutputCorrect(correct_count, y);
    //     hipFree(layer_input[num_layers - 1]);
    //     *scalar_loss = computeLoss(); // Loss Computation
    return;
  }
  /***************************************************************************/
  *scalar_loss = computeLoss();  // Loss Computation

  hipMemGetInfo(&free_bytes, &total_bytes);
  int bef1 = free_bytes;
  hipMalloc(&dlayer_input[num_layers],
             batch_size * num_classes * data_type_size);
  hipMemGetInfo(&free_bytes, &total_bytes);
  int aft1 = free_bytes;
  std::cout << "Allocated to dlayer " << num_layers << ": " << (bef1 - aft1)
            << " free: " << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";

  if (layer_type[num_layers - 1] == SOFTMAX) {
    if (data_type == HIPDNN_DATA_FLOAT) {
      hipMemset(dlayer_input[num_layers], 0,
                 batch_size * num_classes * sizeof(float));
      softmaxLossBackProp<float><<<ceil(1.0 * batch_size / BW), BW>>>(
          this->y, (float *)layer_input[num_layers],
          (float *)dlayer_input[num_layers], batch_size, num_classes,
          softmax_eps);
    } else if (data_type == HIPDNN_DATA_DOUBLE) {
      hipMemset(dlayer_input[num_layers], 0,
                 batch_size * num_classes * sizeof(double));
      softmaxLossBackProp<double><<<ceil(1.0 * batch_size / BW), BW>>>(
          this->y, (double *)layer_input[num_layers],
          (double *)dlayer_input[num_layers], batch_size, num_classes,
          softmax_eps);
    }
  }

  /************************ Backward Propagation starts ***********************/
  std::cout << "Backward Propagation starts: " << '\n';
  for (int i = num_layers - 1; i >= 0; i--) {
    if (i > 0) {
      if (layer_type[i] == ACTV || layer_type[i] == SOFTMAX)
        dlayer_input[i] = dlayer_input[i + 1];

      // Prefetching
      if (offloaded[i - 1]) {
        std::cout << "Prefetching layer " << i - 1 << "\n";
        hipMalloc(&layer_input[i - 1],
                   layer_input_size[i - 1] * data_type_size);
        if (i - 1 != 0) {
          hipMemcpyAsync(layer_input[i - 1], h_layer_input[i - 1],
                          layer_input_size[i - 1] * data_type_size,
                          hipMemcpyHostToDevice, stream_memory);
        } else {
          hipMemcpyAsync(layer_input[i - 1], X,
                          layer_input_size[i - 1] * data_type_size,
                          hipMemcpyHostToDevice, stream_memory);
        }
      }

      hipMemGetInfo(&free_bytes, &total_bytes);
      int bef2 = free_bytes;
      hipMalloc(&dlayer_input[i], layer_input_size[i] * data_type_size);
      hipMemGetInfo(&free_bytes, &total_bytes);
      int aft2 = free_bytes;
      std::cout << "Allocated to dlayer " << i << ": " << (bef2 - aft2)
                << " free: " << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";
    }
    hipMemGetInfo(&free_bytes, &total_bytes);
    std::cout << "BP Before Derivative of Layer " << i << ": "
              << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";
    if (layer_type[i] == CONV) {
      ConvLayerParams *cur_params = (ConvLayerParams *)params[i];

      if (cur_params->activation_mode != ACTIVATION_NONE) {
        checkCUDNN(hipdnnActivationBackward(
            cudnn_handle, cur_params->actv_desc, &alpha,
            cur_params->output_tensor, layer_input[i + 1],
            cur_params->output_tensor, dlayer_input[i + 1],
            cur_params->output_tensor, layer_input[i + 1], &beta,
            cur_params->output_tensor, dlayer_input[i + 1]));
      }

      hipMemGetInfo(&free_bytes, &total_bytes);
      std::cout << "After hipdnnActivationBackward " << i << ": "
                << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";

      size_t temp_data_wksp;

      if (i == 0)
        temp_data_wksp = 0;
      else
        temp_data_wksp = cur_params->bwd_data_workspace_size;

      this->workspace_size =
          max(cur_params->bwd_filter_workspace_size, temp_data_wksp);

      if (i == 1) std::cout << this->workspace_size << "\n";

      std::cout << hipMalloc(&(this->workspace), this->workspace_size) << "\n";

      checkCUDNN(hipdnnConvolutionBackwardBias(
          cudnn_handle, &alpha, cur_params->output_tensor, dlayer_input[i + 1],
          &beta, cur_params->bias_desc, cur_params->db));

      hipMemGetInfo(&free_bytes, &total_bytes);
      std::cout << "After hipdnnConvolutionBackwardBias " << i << ": "
                << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";
      if (this->workspace == NULL) std::cout << "workspace problem\n";

      checkCUDNN(hipdnnConvolutionBackwardFilter(
          cudnn_handle, &alpha, cur_params->input_tensor, layer_input[i],
          cur_params->output_tensor, dlayer_input[i + 1], cur_params->conv_desc,
          cur_params->bwd_filter_algo, this->workspace, this->workspace_size,
          &beta, cur_params->filter_desc, cur_params->dW));

      hipMemGetInfo(&free_bytes, &total_bytes);
      std::cout << "After hipdnnConvolutionBackwardFilter " << i << ": "
                << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";

      if (i > 0)
        checkCUDNN(hipdnnConvolutionBackwardData(
            cudnn_handle, &alpha, cur_params->filter_desc, cur_params->W,
            cur_params->output_tensor, dlayer_input[i + 1],
            cur_params->conv_desc, cur_params->bwd_data_algo, this->workspace,
            workspace_size, &beta, cur_params->input_tensor, dlayer_input[i]));

      hipMemGetInfo(&free_bytes, &total_bytes);
      std::cout << "After hipdnnConvolutionBackwardData " << i << ": "
                << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";

      cur_params->stepParams(cublas_handle, learning_rate);
    }

    else if (layer_type[i] == FULLY_CONNECTED) {
      FCLayerParams *cur_params = (FCLayerParams *)params[i];

      if (cur_params->activation_mode != ACTIVATION_NONE) {
        checkCUDNN(hipdnnActivationBackward(
            cudnn_handle, cur_params->actv_desc, &alpha,
            cur_params->output_tensor, layer_input[i + 1],
            cur_params->output_tensor, dlayer_input[i + 1],
            cur_params->output_tensor, layer_input[i + 1], &beta,
            cur_params->output_tensor, dlayer_input[i + 1]));
      }

      if (data_type == HIPDNN_DATA_FLOAT) {
        // Bias backward
        hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, cur_params->C_out,
                    1, batch_size, &Salpha, (float *)dlayer_input[i + 1],
                    cur_params->C_out, (float *)one_vec, batch_size, &Sbeta,
                    (float *)cur_params->db, cur_params->C_out);

        // Weight backward
        hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, cur_params->C_out,
                    cur_params->C_in, batch_size, &Salpha,
                    (float *)dlayer_input[i + 1], cur_params->C_out,
                    (float *)layer_input[i], cur_params->C_in, &Sbeta,
                    (float *)cur_params->dW, cur_params->C_out);

        // Data backward
        if (i > 0)
          hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, cur_params->C_in,
                      batch_size, cur_params->C_out, &Salpha,
                      (float *)cur_params->W, cur_params->C_out,
                      (float *)dlayer_input[i + 1], cur_params->C_out, &Sbeta,
                      (float *)dlayer_input[i], cur_params->C_in);
      }

      else if (data_type == HIPDNN_DATA_DOUBLE) {
        // Bias backward
        hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, cur_params->C_out,
                    1, batch_size, &Dalpha, (double *)dlayer_input[i + 1],
                    cur_params->C_out, (double *)one_vec, batch_size, &Dbeta,
                    (double *)cur_params->db, cur_params->C_out);

        // Weight backward
        hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, cur_params->C_out,
                    cur_params->C_in, batch_size, &Dalpha,
                    (double *)dlayer_input[i + 1], cur_params->C_out,
                    (double *)layer_input[i], cur_params->C_in, &Dbeta,
                    (double *)cur_params->dW, cur_params->C_out);

        // Data backward
        if (i > 0)
          hipblasDgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, cur_params->C_in,
                      batch_size, cur_params->C_out, &Dalpha,
                      (double *)cur_params->W, cur_params->C_out,
                      (double *)dlayer_input[i + 1], cur_params->C_out, &Dbeta,
                      (double *)dlayer_input[i], cur_params->C_in);
      }
      cur_params->stepParams(cublas_handle, learning_rate);
    }

    else if (layer_type[i] == POOLING) {
      PoolingLayerParams *cur_params = (PoolingLayerParams *)params[i];
      checkCUDNN(
          hipdnnPoolingBackward(cudnn_handle, cur_params->pool_desc, &alpha,
                               cur_params->output_tensor, layer_input[i + 1],
                               cur_params->output_tensor, dlayer_input[i + 1],
                               cur_params->input_tensor, layer_input[i], &beta,
                               cur_params->input_tensor, dlayer_input[i]));
    }

    else if (layer_type[i] == ACTV) {
      ActivationLayerParams *cur_params = (ActivationLayerParams *)params[i];
      checkCUDNN(hipdnnActivationBackward(
          cudnn_handle, cur_params->actv_desc, &alpha, cur_params->input_tensor,
          layer_input[i + 1], cur_params->input_tensor, dlayer_input[i + 1],
          cur_params->input_tensor, layer_input[i], &beta,
          cur_params->input_tensor, dlayer_input[i]));
      continue;
    }

    else if (layer_type[i] == SOFTMAX) {
      SoftmaxLayerParams *cur_params = (SoftmaxLayerParams *)params[i];
      checkCUDNN(hipdnnSoftmaxBackward(
          cudnn_handle, cur_params->algo, cur_params->mode, &alpha,
          cur_params->input_tensor, layer_input[i + 1],
          cur_params->input_tensor, dlayer_input[i + 1], &beta,
          cur_params->input_tensor, dlayer_input[i]));
      continue;
    }

    hipMemGetInfo(&free_bytes, &total_bytes);
    std::cout << "Before Synchronization " << i << ": "
              << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";
    hipStreamSynchronize(stream_compute);

    hipMemGetInfo(&free_bytes, &total_bytes);
    std::cout << "After Synchronization " << i << ": "
              << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";
    hipStreamSynchronize(stream_memory);

    if (layer_type[i] == CONV) hipFree(this->workspace);

    hipMemGetInfo(&free_bytes, &total_bytes);
    std::cout << "BP After Derivative of Layer " << i << ": "
              << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";

    hipMemGetInfo(&free_bytes, &total_bytes);
    int bef3 = free_bytes;
    hipFree(layer_input[i + 1]);
    hipMemGetInfo(&free_bytes, &total_bytes);
    int aft3 = free_bytes;
    std::cout << "freed to layer " << i + 1 << ": " << (aft3 - bef3)
              << " free: " << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";

    hipMemGetInfo(&free_bytes, &total_bytes);
    int bef4 = free_bytes;
    hipFree(dlayer_input[i + 1]);
    hipMemGetInfo(&free_bytes, &total_bytes);
    int aft4 = free_bytes;
    std::cout << "freed to dlayer " << i + 1 << ": " << (aft4 - bef4)
              << " free: " << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";

    if (i == 0) {
      hipFree(layer_input[i]);
      hipMemGetInfo(&free_bytes, &total_bytes);
      int aft5 = free_bytes;
      std::cout << "freed to layer " << i << ": " << (aft5 - aft4)
                << " free: " << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";
      hipFree(dlayer_input[i]);
      hipMemGetInfo(&free_bytes, &total_bytes);
      int aft6 = free_bytes;
      std::cout << "freed to layer " << i << ": " << (aft6 - aft5)
                << " free: " << free_bytes / (1024.0 * 1024.0 * 1024.0) << "\n";
    }

    hipMemGetInfo(&free_bytes, &total_bytes);
    std::cout << "freed up feature map and its derivative after layer " << i
              << " of BP: " << free_bytes / (1024.0 * 1024.0 * 1024.0) << '\n';
  }
  std::cout << "Backward Propagation ends: " << '\n';
  /************************ Backward Propagation ends ***********************/

  hipMemGetInfo(&free_bytes, &total_bytes);
  std::cout << "free mem before final free: "
            << free_bytes / (1024.0 * 1024.0 * 1024.0) << '\n';
  for (int k = 0; k < num_layers; k++) {
    if (layer_input[k] != NULL) hipFree(layer_input[k]);
    if (dlayer_input[k] != NULL) hipFree(dlayer_input[k]);
  }
  hipMemGetInfo(&free_bytes, &total_bytes);
  std::cout << "free mem after 1FP1BP: "
            << free_bytes / (1024.0 * 1024.0 * 1024.0) << '\n';

  // Make offloaded array to all false
  for (int c = 0; c < num_layers; c++) offloaded[c] = false;
}
